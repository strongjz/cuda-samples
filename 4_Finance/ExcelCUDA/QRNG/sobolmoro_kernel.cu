#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2013 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * Portions Copyright (c) 1993-2013 NVIDIA Corporation.  All rights reserved.
 * Portions Copyright (c) 2009 Mike Giles, Oxford University.  All rights reserved.
 * Portions Copyright (c) 2008 Frances Y. Kuo and Stephen Joe.  All rights reserved.
 *
 * Sobol Quasi-random Number Generator example
 *
 * Based on CUDA code submitted by Mike Giles, Oxford University, United Kingdom
 * http://people.maths.ox.ac.uk/~gilesm/
 *
 * and C code developed by Stephen Joe, University of Waikato, New Zealand
 * and Frances Kuo, University of New South Wales, Australia
 * http://web.maths.unsw.edu.au/~fkuo/sobol/
 *
 * For theoretical background see:
 *
 * P. Bratley and B.L. Fox.
 * Implementing Sobol's quasirandom sequence generator
 * http://portal.acm.org/citation.cfm?id=42288
 * ACM Trans. on Math. Software, 14(1):88-100, 1988
 *
 * S. Joe and F. Kuo.
 * Remark on algorithm 659: implementing Sobol's quasirandom sequence generator.
 * http://portal.acm.org/citation.cfm?id=641879
 * ACM Trans. on Math. Software, 29(1):49-57, 2003
 *
 */

#include <hip/hip_runtime.h>
#include "sobol.h"
#include "sobolmoro_kernel.h"

#define k_2powneg32 2.3283064E-10F

__device__ inline float MoroInvCND(float P)
{
    const float a1 = 2.50662823884f;
    const float a2 = -18.61500062529f;
    const float a3 = 41.39119773534f;
    const float a4 = -25.44106049637f;
    const float b1 = -8.4735109309f;
    const float b2 = 23.08336743743f;
    const float b3 = -21.06224101826f;
    const float b4 = 3.13082909833f;
    const float c1 = 0.337475482272615f;
    const float c2 = 0.976169019091719f;
    const float c3 = 0.160797971491821f;
    const float c4 = 2.76438810333863E-02f;
    const float c5 = 3.8405729373609E-03f;
    const float c6 = 3.951896511919E-04f;
    const float c7 = 3.21767881768E-05f;
    const float c8 = 2.888167364E-07f;
    const float c9 = 3.960315187E-07f;
    float y;
    float z;

    if (P <= 0 || P >= 1.0f)
        return __int_as_float(0x7FFFFFFF);

    y = P - 0.5f;

    if (fabsf(y) < 0.42f)
    {
        z = y * y;
        z = y * (((a4 * z + a3) * z + a2) * z + a1) / ((((b4 * z + b3) * z + b2) * z + b1) * z + 1.0f);
    }
    else
    {
        if (y > 0)
            z = __logf(-__logf(1.0f - P));
        else
            z = __logf(-__logf(P));

        z = c1 + z * (c2 + z * (c3 + z * (c4 + z * (c5 + z * (c6 + z * (c7 + z * (c8 + z * c9)))))));

        if (y < 0)
            z = -z;
    }

    return z;
}

__global__ void sobolmoro_kernel(unsigned n_vectors, unsigned n_dimensions, unsigned *d_directions, float *d_output)
{
    __shared__ unsigned int v[n_directions];

    // Offset into the correct dimension as specified by the
    // block y coordinate
    d_directions = d_directions + n_directions * blockIdx.y;
    d_output = d_output +  n_vectors * blockIdx.y;

    // Copy the direction numbers for this dimension into shared
    // memory - there are only 32 direction numbers so only the
    // first 32 (n_directions) threads need participate.
    if (threadIdx.x < n_directions)
    {
        v[threadIdx.x] = d_directions[threadIdx.x];
    }

    __syncthreads();

    // Set initial index (i.e. which vector this thread is
    // computing first) and stride (i.e. step to the next vector
    // for this thread)
    int i0     = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;

    // Get the gray code of the index
    // c.f. Numerical Recipes in C, chapter 20
    // http://www.nrbook.com/a/bookcpdf/c20-2.pdf
    unsigned int g = (i0 + 1) ^ ((i0 + 1) >> 1);

    // Initialisation for first point x[i0]
    // In the Bratley and Fox paper this is equation (*), where
    // we are computing the value for x[n] without knowing the
    // value of x[n-1].
    unsigned int X = 0;
    unsigned int mask;

    for (unsigned int k = 0 ; k < __ffs(stride) - 1 ; k++)
    {
        // We want X ^= g_k * v[k], where g_k is one or zero.
        // We do this by setting a mask with all bits equal to
        // g_k. In reality we keep shifting g so that g_k is the
        // LSB of g. This way we avoid multiplication.
        mask = - (g & 1);
        X ^= mask & v[k];
        g = g >> 1;
    }

    if (i0 < n_vectors)
    {
        d_output[i0] = MoroInvCND((float)X * k_2powneg32);
    }

    // Now do rest of points, using the stride
    // Here we want to generate x[i] from x[i-stride] where we
    // don't have any of the x in between, therefore we have to
    // revisit the equation (**), this is easiest with an example
    // so assume stride is 16.
    // From x[n] to x[n+16] there will be:
    //   8 changes in the first bit
    //   4 changes in the second bit
    //   2 changes in the third bit
    //   1 change in the fourth
    //   1 change in one of the remaining bits
    //
    // What this means is that in the equation:
    //   x[n+1] = x[n] ^ v[p]
    //   x[n+2] = x[n+1] ^ v[q] = x[n] ^ v[p] ^ v[q]
    //   ...
    // We will apply xor with v[1] eight times, v[2] four times,
    // v[3] twice, v[4] once and one other direction number once.
    // Since two xors cancel out, we can skip even applications
    // and just apply xor with v[4] (i.e. log2(16)) and with
    // the current applicable direction number.
    // Note that all these indices count from 1, so we need to
    // subtract 1 from them all to account for C arrays counting
    // from zero.
    unsigned int v_log2stridem1 = v[__ffs(stride) - 2];
    unsigned int v_stridemask = stride - 1;

    for (unsigned int i = i0 + stride ; i < n_vectors ; i += stride)
    {
        // x[i] = x[i-stride] ^ v[b] ^ v[c]
        //  where b is log2(stride) minus 1 for C array indexing
        //  where c is the index of the rightmost zero bit in i,
        //  not including the bottom log2(stride) bits, minus 1
        //  for C array indexing
        // In the Bratley and Fox paper this is equation (**)
        X ^= v_log2stridem1 ^ v[__ffs(~((i - stride) | v_stridemask)) - 1];
        d_output[i] = MoroInvCND((float)X * k_2powneg32);
    }
}
